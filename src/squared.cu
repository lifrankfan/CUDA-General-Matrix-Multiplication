#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "squared.cuh";
//#include <hip/device_functions.h>

#define TILE_WIDTH 16

__global__ void squaredKernel(float* A, float* B, float* C, int width) {
    // set size of subtiles
    __shared__ float subA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float subB[TILE_WIDTH][TILE_WIDTH];

    // convenience
    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;

    // because squared, only require row
    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;

    // temporary value for each subtile
    float Cvalue = 0.0;

    for (int m = 0; m < (width - 1) / TILE_WIDTH + 1; m++) {
        if (row < width && m * TILE_WIDTH + tx < width) {
            subA[ty][tx] = A[row * width + m * TILE_WIDTH + tx];
        }
        else {
            subA[ty][tx] = 0.0;
        }

        if (m * TILE_WIDTH + ty < width && col < width) {
            subB[ty][tx] = B[(m * TILE_WIDTH + ty) * width + col];
        }
        else {
            subB[ty][tx] = 0.0;
        }

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; k++) {
            Cvalue += subA[ty][k] * subB[k][tx];
        }

        __syncthreads();
    }

    // completes one cell entry
    if (row < width && col < width) {
        C[row * width + col] = Cvalue;
    }
}

int squared(FILE * file_A, FILE * file_B) {
/*==========================================================*/
// Set row and column
    int rows_A, cols_A, rows_B, cols_B;

    // reads first line of files as dimensions of matrix
    fscanf(file_A, "%d %d", &rows_A, &cols_A);
    fscanf(file_B, "%d %d", &rows_B, &cols_B);
/*===========================================================*/

    // ensure matrices are squared
    if (cols_A != rows_B) {
        fprintf(stderr, "Error: incompatible matrix dimensions.\n");
        exit(EXIT_FAILURE);
    }

    int width = cols_B;

    // allocate memory for host matrices
    float* h_A = (float*)malloc(rows_A * cols_A * sizeof(float));
    float* h_B = (float*)malloc(rows_B * cols_B * sizeof(float));
    float* h_C = (float*)malloc(rows_A * cols_B * sizeof(float));

    // read data into h_A and h_B
    for (int i = 0; i < rows_A; i++) {
        for (int j = 0; j < cols_A; j++) {
            fscanf(file_A, "%f", &h_A[i * cols_A + j]);
        }
    }

    for (int i = 0; i < rows_B; i++) {
        for (int j = 0; j < cols_B; j++) {
            fscanf(file_B, "%f", &h_B[i * cols_B + j]);
        }
    }

    // allocate GPU memory
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, rows_A * cols_A * sizeof(float));
    hipMalloc((void**)&d_B, rows_B * cols_B * sizeof(float));
    hipMalloc((void**)&d_C, rows_A * cols_B * sizeof(float));

    // copy memory to GPU
    hipMemcpy(d_A, h_A, rows_A * cols_A * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, rows_B * cols_B * sizeof(float), hipMemcpyHostToDevice);

    // grid and block dimensions
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((width - 1) / TILE_WIDTH + 1, (rows_A - 1) / TILE_WIDTH + 1);

    // launch the kernel
    squaredKernel <<<dimGrid, dimBlock>>> (d_A, d_B, d_C, width);

    // copy GPU memory back to CPU once the kernel completes
    hipMemcpy(h_C, d_C, rows_A * cols_B * sizeof(float), hipMemcpyDeviceToHost);

    // prints resulting matrix
    for (int i = 0; i < rows_A; i++) {
        for (int j = 0; j < cols_B; j++) {
            printf("%.2f ", h_C[i * cols_B + j]);
        }
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    fclose(file_A);
    fclose(file_B);

    return 1;
}
